
#include <hip/hip_runtime.h>
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#define BLOCK_SIZE 512

__global__ void reduction(float *out, float *in, unsigned size)
{
    /********************************************************************
    Load a segment of the input vector into shared memory
    Traverse the reduction tree
    Write the computed sum to the output vector at the correct index
    ********************************************************************/

    // INSERT KERNEL CODE HERE
    
	__shared__ float pSum[2 * BLOCK_SIZE];
	
	int i = 2 * blockIdx.x * blockDim.x;
	int tx = threadIdx.x;
	
	pSum[tx] = (tx < size) ? in[i + tx] : 0;
	pSum[blockDim.x + tx] = ((blockDim.x + tx) < size) ? in[i + blockDim.x + tx] : 0;
	
	for(int offset = blockDim.x; offset >= 1; offset >>= 1) {
		__syncthreads();
		if(tx < offset)
			pSum[tx] += pSum[tx + offset];
	}
	
	if(tx == 0) {
		out[blockIdx.x + tx] = pSum[tx];
	}
}